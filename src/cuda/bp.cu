#include "cuda/bp.h"
#include "cuda/util.h"

#include "cuda/core.h"

namespace infer {
namespace cuda {

bp::bp(const crf &crf)
    : method(crf)
    , current_iteration(0)
    , dev_l_(0), dev_r_(0), dev_u_(0), dev_d_(0) {

    const size_t size = crf.width_ * crf.height_ * crf.labels_ * sizeof(float);
    cuda_check(hipMalloc(&dev_l_, size));
    cuda_check(hipMalloc(&dev_r_, size));
    cuda_check(hipMalloc(&dev_u_, size));
    cuda_check(hipMalloc(&dev_d_, size));

    cuda_check(hipMemset(dev_l_, 0, size));
    cuda_check(hipMemset(dev_r_, 0, size));
    cuda_check(hipMemset(dev_u_, 0, size));
    cuda_check(hipMemset(dev_d_, 0, size));
}

void bp::run(const unsigned iterations) {
    if (iterations != 0) {
        dirty_ = true;
    }

    dim3 block(8, 16); // only half of the pixels are updated because of the checkboard pattern
    dim3 grid(((crf_.width_ + 1) / 2 + block.x - 1) / block.x, (crf_.height_ + block.y - 1) / block.y);

    for (unsigned i = 0; i < iterations; ++i) {
        ++current_iteration;

        trbp_run<<<grid, block>>>(crf_.labels_, crf_.width_, crf_.height_, current_iteration, crf_.type_, crf_.lambda_, crf_.trunc_, crf_.dev_pairwise_, dev_l_, dev_r_, dev_u_, dev_d_, crf_.dev_unary_, 0);
        cuda_check(hipGetLastError());
    }
}

void bp::update_dev_result() const {
    dim3 block(16, 16);
    dim3 grid((crf_.width_ + block.x - 1) / block.x, (crf_.height_ + block.y - 1) / block.y);
    trbp_get_results<<<grid, block>>>(crf_.labels_, crf_.width_, crf_.height_, dev_l_, dev_r_, dev_u_, dev_d_, crf_.dev_unary_, dev_result_, 0);
    cuda_check(hipGetLastError());
}

std::string bp::get_name() const {
    return "gpu_bp";
}

bp::~bp() {
    hipFree(dev_l_);
    hipFree(dev_r_);
    hipFree(dev_u_);
    hipFree(dev_d_);
}

}
}
