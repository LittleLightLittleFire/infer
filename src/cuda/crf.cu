#include "hip/hip_runtime.h"
#include "cuda/crf.h"
#include "cuda/util.h"

#include <hip/hip_runtime.h>

namespace infer {
namespace cuda {

crf::crf(const unsigned width, const unsigned height, const unsigned labels, const std::vector<float> unary, const float lambda, const unsigned norm, const unsigned trunc)
    : width_(width)
    , height_(height)
    , labels_(labels)
    , dev_unary_(0)
    , lambda_(lambda)
    , type_(norm == 1 ? L1 : L2)
    , trunc_(trunc)
    , dev_pairwise_(0) {

    cuda_check(hipMalloc(&dev_unary_, width * height * labels * sizeof(float)));
    cuda_check(hipMemcpy(dev_unary_, &unary[0], width * height * labels * sizeof(float), hipMemcpyHostToDevice));
}

crf::crf(const unsigned width, const unsigned height, const unsigned labels, const std::vector<float> unary, const float lambda, const std::vector<float> pairwise)
    : width_(width)
    , height_(height)
    , labels_(labels)
    , dev_unary_(0)
    , lambda_(lambda)
    , type_(ARRAY)
    , trunc_(0)
    , dev_pairwise_(0) {

    cuda_check(hipMalloc(&dev_unary_, width * height * labels * sizeof(float)));
    cuda_check(hipMalloc(&dev_pairwise_, width * height * labels * labels * 2 * sizeof(float)));

    cuda_check(hipMemcpy(dev_unary_, &unary[0], width * height * labels * sizeof(float), hipMemcpyHostToDevice));
    cuda_check(hipMemcpy(dev_pairwise_, &pairwise[0], width * height * labels * labels * 2 * sizeof(float), hipMemcpyHostToDevice));
}

crf::~crf() {
    hipFree(dev_unary_);

    if (dev_pairwise_) {
        hipFree(dev_pairwise_);
    }
}

}
}
