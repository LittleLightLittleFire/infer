#include "hip/hip_runtime.h"
#include "cuda/crf.h"

#include <hip/hip_runtime.h>

namespace infer {
namespace cuda {

crf::crf(const unsigned width, const unsigned height, const unsigned labels, const std::vector<float> unary, const float lambda, const unsigned norm, const unsigned trunc)
    : width_(width)
    , height_(height)
    , labels_(labels)
    , dev_unary_(0)
    , lambda_(lambda)
    , type_(norm == 1 ? L1 : L2)
    , trunc_(trunc)
    , dev_pairwise_(0) {

    // TODO:
}

crf::crf(const unsigned width, const unsigned height, const unsigned labels, const std::vector<float> unary, const float lambda, const std::vector<float> pairwise)
    : width_(width)
    , height_(height)
    , labels_(labels)
    , dev_unary_(0)
    , lambda_(lambda)
    , type_(ARRAY)
    , trunc_(0)
    , dev_pairwise_(0) {

    // TODO:
}

crf::~crf() {
    // TODO:
}

}
}

#endif // INFER_CUDA_CRF_H
