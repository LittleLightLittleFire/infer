#include "cuda/util.h"

#include <sstream>

namespace infer {
namespace cuda {

cuda_exception::cuda_exception(std::string err)
    : runtime_error(err) {

}

void cuda_throw(const char *file, const unsigned line, const hipError_t err) {
    if (err != hipSuccess) {
        std::stringstream ss;
        ss << file << ":" << line <<  " " << hipGetErrorString(err);
        throw cuda_exception(ss.str());
    }
}

}
}

