#include "hip/hip_runtime.h"
#include "cuda/core.h"

#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

namespace infer {
namespace cuda {

namespace {
__device__ void fast_L1_helper(const unsigned labels, const float curr_min, const float s, const float t, float *out) {

    // do the O(n) trick
    for (unsigned i = 1; i < labels; ++i) {
        out[i] = fminf(out[i-1] + s, out[i]);
    }

    for (unsigned i = labels - 1; i-- > 0; ) {
        out[i] = fminf(out[i+1] + s, out[i]);
    }

    // truncate
    for (unsigned i = 0; i < labels; ++i) {
        out[i] = fminf(curr_min + t, out[i]);
    }

    // normalise
    const float val = out[0];
    for (unsigned i = 0; i < labels; ++i) {
        out[i] -= val;
    }
}

__device__ void send_message_L1(const unsigned labels, const unsigned x, const unsigned y, const float lambda, const float trunc, const float *m1, const float *m2, const float *m3, const float *pot, float *out) {
    float curr_min = HIP_MAX_NORMAL_F;

    // add all the incoming messages together
    for (unsigned i = 0; i < labels; ++i) {
        out[i] = m1[i] + m2[i] + m3[i] + pot[i];
        curr_min = fminf(curr_min, out[i]);
    }

    // adjust lambda because of trbp
    fast_L1_helper(labels, curr_min, lambda, lambda * trunc, out);
}

__device__ void send_message_L1(const unsigned labels, const unsigned x, const unsigned y, const float lambda, const float trunc
                               , const float *m1, const float *m2, const float *m3, const float *opp
                               , const float rm1, const float rm2, const float rm3, const float ropp
                               , const float *pot, float *out) {

    float curr_min = HIP_MAX_NORMAL_F;

    // add all the incoming messages together
    for (unsigned i = 0; i < labels; ++i) {
        out[i] = pot[i] + m1[i] * rm1 + m2[i] * rm2 + m3[i] * rm3 - opp[i] * (1 - ropp);
        curr_min = fminf(curr_min, out[i]);
    }

    // adjust lambda because of trbp
    fast_L1_helper(labels, curr_min, lambda * (1 / ropp), lambda * trunc * (1 / ropp), out);
}

}

__global__ void trbp_run(const unsigned labels, const unsigned w, const unsigned h, const unsigned i, const crf::type type, const float lambda, const float trunc, const float *pairwise, float *l, float *r, float *u, float *d, const float *pot, const float *rho) {
    const unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;

    const unsigned x = ix * 2 + ((i + iy) % 2 == 0 ? 1 : 0);
    const unsigned y = iy;

    // bounds check
    if (x == 0 || y == 0 || x >= w - 1 || y >= h - 1) {
        return;
    }

    //printf("thread (%u, %u), block (%u, %u) %u %u\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, x, y);

    const float up = rho[(w * y + x) * 2];
    const float left = rho[(w * y + x) * 2 + 1];
    const float down = rho[(w * (y - 1) + x) * 2];
    const float right = rho[(w * y + (x - 1)) * 2 + 1];

    const unsigned base = (w * y + x) * labels;
    switch (type) {
        case crf::L1:
            if (rho) {
                //                                           m1        m2        m3        opp
                send_message_L1(labels, x, y, lambda, trunc, u + base, d + base, r + base, l + base
                                                           , up      , down    , right   , left     , pot + base, r + (w * y + x + 1) * labels);

                send_message_L1(labels, x, y, lambda, trunc, u + base, d + base, l + base, r + base
                                                           , up      , down    , left    , right    , pot + base, l + (w * y + x - 1) * labels);

                send_message_L1(labels, x, y, lambda, trunc, d + base, l + base, r + base, u + base
                                                           , down    , left    , right   , up       , pot + base, d + (w * (y + 1) + x) * labels);

                send_message_L1(labels, x, y, lambda, trunc, u + base, l + base, r + base, d + base
                                                           , up      , left    , right   , down     , pot + base, u + (w * (y - 1) + x) * labels);
            } else {
                send_message_L1(labels, x, y, lambda, trunc, u + base, d + base, r + base, pot + base, r + (w * y + x + 1) * labels);
                send_message_L1(labels, x, y, lambda, trunc, u + base, d + base, l + base, pot + base, l + (w * y + x - 1) * labels);
                send_message_L1(labels, x, y, lambda, trunc, d + base, l + base, r + base, pot + base, d + (w * (y + 1) + x) * labels);
                send_message_L1(labels, x, y, lambda, trunc, u + base, l + base, r + base, pot + base, u + (w * (y - 1) + x) * labels);
            }
            break;
        case crf::L2: // TODO:
            break;
        case crf::ARRAY: // TODO:
            break;
    }
}

__global__ void trbp_get_results(const unsigned labels, const unsigned w, const unsigned h, const float *l, const float *r, const float *u, const float *d, const float *pot, unsigned *out, const float *rho) {
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

    // bounds check
    if (x >= w || y >= h) {
        return;
    }

    unsigned min_label = 0;
    float min_value = HIP_MAX_NORMAL_F;

    const unsigned base = (w * y + x) * labels;

    for (uint i = 0; i < labels; ++i) {
        float val = (pot + base)[i];

        if (rho) {
            if (y != h - 1) val += (u + base)[i] * rho[(w * y + x) * 2];
            if (x != w - 1) val += (l + base)[i] * rho[(w * y + x) * 2 + 1];
            if (y != 0)     val += (d + base)[i] * rho[(w * (y - 1) + x) * 2];
            if (x != 0)     val += (r + base)[i] * rho[(w * y + (x - 1)) * 2 + 1];
        } else {
            val += (l + base)[i] + (r + base)[i] + (u + base)[i] + (d + base)[i];
        }

        if (val < min_value) {
            min_label = i;
            min_value = val;
        }
    }

    out[x + y * w] = min_label;
}

}
}
