#include "hip/hip_runtime.h"
#include "cuda/core.h"

#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

namespace infer {
namespace cuda {

namespace {
__device__ void fast_L1_helper(const unsigned labels, const float curr_min, const float s, const float t, float *out) {

    // do the O(n) trick
    for (unsigned i = 1; i < labels; ++i) {
        out[i] = fminf(out[i-1] + s, out[i]);
    }

    for (unsigned i = labels - 1; i-- > 0; ) {
        out[i] = fminf(out[i+1] + s, out[i]);
    }

    // truncate
    for (unsigned i = 0; i < labels; ++i) {
        out[i] = fminf(curr_min + t, out[i]);
    }

    // normalise
    const float val = out[0];
    for (unsigned i = 0; i < labels; ++i) {
        out[i] -= val;
    }
}

__device__ void send_message_L1(const unsigned labels, const unsigned x, const unsigned y, const float lambda, const float trunc, const float *m1, const float *m2, const float *m3, const float *pot, float *out) {
    float curr_min = HIP_MAX_NORMAL_F;

    // add all the incoming messages together
    for (unsigned i = 0; i < labels; ++i) {
        out[i] = m1[i] + m2[i] + m3[i] + pot[i];
        curr_min = fminf(curr_min, out[i]);
    }

    // adjust lambda because of trbp
    fast_L1_helper(labels, curr_min, lambda, lambda * trunc, out);
}

__device__ void send_message_L1(const unsigned labels, const unsigned x, const unsigned y, const float lambda, const float trunc
                               , const float *m1, const float *m2, const float *m3, const float *opp
                               , const float rm1, const float rm2, const float rm3, const float ropp
                               , const float *pot, float *out) {

    float curr_min = HIP_MAX_NORMAL_F;

    // add all the incoming messages together
    for (unsigned i = 0; i < labels; ++i) {
        out[i] = pot[i] + m1[i] * rm1 + m2[i] * rm2 + m3[i] * rm3 - opp[i] * (1 - ropp);
        curr_min = fminf(curr_min, out[i]);
    }

    // adjust lambda because of trbp
    fast_L1_helper(labels, curr_min, lambda * (1 / ropp), lambda * trunc * (1 / ropp), out);
}

__device__ float *ndx(const unsigned labels, const unsigned width, float *dir, const unsigned x, const unsigned y) {
    return labels * (x + y * width) + dir;
}

__device__ const float *cndx(const unsigned labels, const unsigned width, const float *dir, const unsigned x, const unsigned y) {
    return labels * (x + y * width) + dir;
}

}

/** initalise messages using the messages from the layer below */
__global__ void prime(const unsigned lbl, const unsigned w, const unsigned h, const unsigned prev_w, const float *prev_msg, float *out) {
    const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    // boundary check
    if (x >= w || y >= h) {
        return;
    }

    // initaise to the last layer's (x/2, y/2)
    float *target = ndx(lbl, w, out, x, y);
    const float *source = cndx(lbl, prev_w, prev_msg, x / 2, y / 2);

    for (unsigned i = 0; i < lbl; ++i) {
        target[i] = source[i];
    }
}


/** generate the next layer's potentials */
__global__ void fill_next_layer_pot(const unsigned labels, const unsigned width, const unsigned height, const unsigned max_width, const unsigned max_height, const float *pot, float *out) {
    const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    // bounds check
    if (x >= width || y >= height) {
        return;
    }

    // collapse the potential in a 2x2 area
    float *target = ndx(labels, width, out, x, y);
    const float *top_left = cndx(labels, max_width, pot, 2 * x, 2 * y);;

    for (unsigned i = 0; i < labels; ++i) {
        target[i] = top_left[i];
    }

    if (2 * x + 1 < max_width) {
        const float *top_right = cndx(labels, max_width, pot, 2 * x + 1, 2 * y);;
        for (unsigned i = 0; i < labels; ++i) {
            target[i] += top_right[i];
        }
    }

    if (2 * (y + 1) < max_height) {
        const float *bottom_left = cndx(labels, max_width, pot, 2 * x, 2 * (y + 1));
        for (unsigned i = 0; i < labels; ++i) {
            target[i] += bottom_left[i];
        }
    }

    if (2 * x + 1 < max_width && 2 * (y + 1) < max_height) {
        const float *bottom_right = cndx(labels, max_width, pot, 2 * x + 1, 2 * (y + 1));
        for (unsigned i = 0; i < labels; ++i) {
            target[i] += bottom_right[i];
        }
    }
}

__global__ void trbp_run(const unsigned labels, const unsigned w, const unsigned h, const unsigned i, const crf::type type, const float lambda, const float trunc, const float *pairwise, float *l, float *r, float *u, float *d, const float *pot, const float *rho) {
    const unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;

    const unsigned x = ix * 2 + ((i + iy) % 2 == 0 ? 1 : 0);
    const unsigned y = iy;

    // bounds check
    if (x == 0 || y == 0 || x >= w - 1 || y >= h - 1) {
        return;
    }

    //printf("thread (%u, %u), block (%u, %u) %u %u\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, x, y);

    const float up = rho[(w * y + x) * 2];
    const float left = rho[(w * y + x) * 2 + 1];
    const float down = rho[(w * (y - 1) + x) * 2];
    const float right = rho[(w * y + (x - 1)) * 2 + 1];

    const unsigned base = (w * y + x) * labels;
    switch (type) {
        case crf::L1:
            if (rho) {
                //                                           m1        m2        m3        opp
                send_message_L1(labels, x, y, lambda, trunc, u + base, d + base, r + base, l + base
                                                           , up      , down    , right   , left     , pot + base, r + (w * y + x + 1) * labels);

                send_message_L1(labels, x, y, lambda, trunc, u + base, d + base, l + base, r + base
                                                           , up      , down    , left    , right    , pot + base, l + (w * y + x - 1) * labels);

                send_message_L1(labels, x, y, lambda, trunc, d + base, l + base, r + base, u + base
                                                           , down    , left    , right   , up       , pot + base, d + (w * (y + 1) + x) * labels);

                send_message_L1(labels, x, y, lambda, trunc, u + base, l + base, r + base, d + base
                                                           , up      , left    , right   , down     , pot + base, u + (w * (y - 1) + x) * labels);
            } else {
                send_message_L1(labels, x, y, lambda, trunc, u + base, d + base, r + base, pot + base, r + (w * y + x + 1) * labels);
                send_message_L1(labels, x, y, lambda, trunc, u + base, d + base, l + base, pot + base, l + (w * y + x - 1) * labels);
                send_message_L1(labels, x, y, lambda, trunc, d + base, l + base, r + base, pot + base, d + (w * (y + 1) + x) * labels);
                send_message_L1(labels, x, y, lambda, trunc, u + base, l + base, r + base, pot + base, u + (w * (y - 1) + x) * labels);
            }
            break;
        case crf::L2: // TODO:
            break;
        case crf::ARRAY: // TODO:
            break;
    }
}

__global__ void trbp_get_results(const unsigned labels, const unsigned w, const unsigned h, const float *l, const float *r, const float *u, const float *d, const float *pot, unsigned *out, const float *rho) {
    const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    // bounds check
    if (x >= w || y >= h) {
        return;
    }

    unsigned min_label = 0;
    float min_value = HIP_MAX_NORMAL_F;

    const unsigned base = (w * y + x) * labels;

    for (unsigned i = 0; i < labels; ++i) {
        float val = (pot + base)[i];

        if (rho) {
            if (y != h - 1) val += (u + base)[i] * rho[(w * y + x) * 2];
            if (x != w - 1) val += (l + base)[i] * rho[(w * y + x) * 2 + 1];
            if (y != 0)     val += (d + base)[i] * rho[(w * (y - 1) + x) * 2];
            if (x != 0)     val += (r + base)[i] * rho[(w * y + (x - 1)) * 2 + 1];
        } else {
            val += (l + base)[i] + (r + base)[i] + (u + base)[i] + (d + base)[i];
        }

        if (val < min_value) {
            min_label = i;
            min_value = val;
        }
    }

    out[x + y * w] = min_label;
}

}
}
