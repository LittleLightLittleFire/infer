#include <limits>
#include <vector>
#include <iostream>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

namespace {
    typedef unsigned char uchar;
    typedef unsigned uint;

    __device__ float *ndx(const uint labels, const uint width, float *dir, const uint x, const uint y) {
        return labels * (x + y * width) + dir;
    }

    __device__ const float *cndx(const uint labels, const uint width, const float *dir, const uint x, const uint y) {
        return labels * (x + y * width) + dir;
    }

    /** generate the next layer's potentials */
    __global__ void fill_next_layer_pot(const uint labels, const uint width, const uint height, const uint max_width, const uint max_height, const float *pot, float *out) {
        const uint x = blockIdx.x * blockDim.x + threadIdx.x;
        const uint y = blockIdx.y * blockDim.y + threadIdx.y;

        // bounds check
        if (x >= width || y >= height) {
            return;
        }

        // collapse the potential in a 2x2 area
        float *target = ndx(labels, width, out, x, y);
        const float *top_left = cndx(labels, max_width, pot, 2 * x, 2 * y);;

        for (uint i = 0; i < labels; ++i) {
            target[i] = top_left[i];
        }

        if (2 * x + 1 < max_width) {
            const float *top_right = cndx(labels, max_width, pot, 2 * x + 1, 2 * y);;
            for (uint i = 0; i < labels; ++i) {
                target[i] += top_right[i];
            }
        }

        if (2 * (y + 1) < max_height) {
            const float *bottom_left = cndx(labels, max_width, pot, 2 * x, 2 * (y + 1));
            for (uint i = 0; i < labels; ++i) {
                target[i] += bottom_left[i];
            }
        }

        if (2 * x + 1 < max_width && 2 * (y + 1) < max_height) {
            const float *bottom_right = cndx(labels, max_width, pot, 2 * x + 1, 2 * (y + 1));
            for (uint i = 0; i < labels; ++i) {
                target[i] += bottom_right[i];
            }
        }
    }

    /** max product send message */
    __device__ void send_msg_map(const uint labels, const float disc_trunc, const float *m1, const float *m2, const float *m3, const float *pot, float *out) {
        float curr_min = HIP_MAX_NORMAL_F;

        // add all the incoming messages together
        for (uint i = 0; i < labels; ++i) {
            out[i] = m1[i] + m2[i] + m3[i] + pot[i];
            curr_min = fminf(curr_min, out[i]);
        }

        // do the O(n) trick
        for (uint i = 1; i < labels; ++i) {
            out[i] = fminf(out[i-1] + 1.0, out[i]);
        }

        for (int i = labels - 2; i >= 0; --i) {
            out[i] = fminf(out[i+1] + 1.0, out[i]);
        }

        // truncate
        for (uint i = 0; i < labels; ++i) {
            out[i] = fminf(curr_min + disc_trunc, out[i]);
        }

        // normalise
        float sum = 0;
        for (uint i = 0; i < labels; ++i) {
            sum += out[i];
        }

        sum /= static_cast<float>(labels);
        for (uint i = 0; i < labels; ++i) {
            out[i] -= sum;
        }
    }

    /** loopy belief propagation */
    __global__ void bp(const uint lbl, const uint w, const uint h, const float disc_trunc, const uint i, const float *pot, float *u, float *d, float *l, float *r) {
        const uint x = blockIdx.x * blockDim.x + threadIdx.x;
        const uint y = blockIdx.y * blockDim.y + threadIdx.y;

        // bounds check
        if (x < 1 || y < 1 || x >= w - 1|| y >= h - 1) {
            return;
        }

        // check if this thread is active for this iteration
        if ((x + y + i) % 2 == 0) {
            send_msg_map(lbl, disc_trunc, cndx(lbl, w, u, x, y+1), cndx(lbl, w, l , x+1, y), cndx(lbl, w, r, x-1, y), cndx(lbl, w, pot, x, y), ndx(lbl, w, u, x, y));
            send_msg_map(lbl, disc_trunc, cndx(lbl, w, d, x, y-1), cndx(lbl, w, l , x+1, y), cndx(lbl, w, r, x-1, y), cndx(lbl, w, pot, x, y), ndx(lbl, w, d, x, y));
            send_msg_map(lbl, disc_trunc, cndx(lbl, w, u, x, y+1), cndx(lbl, w, d, x, y-1),  cndx(lbl, w, r, x-1, y), cndx(lbl, w, pot, x, y), ndx(lbl, w, r, x, y));
            send_msg_map(lbl, disc_trunc, cndx(lbl, w, u, x, y+1), cndx(lbl, w, d, x, y-1),  cndx(lbl, w, l, x+1, y), cndx(lbl, w, pot, x, y), ndx(lbl, w, l, x, y));
        }
    }

    /** initalise messages using the messages from the layer below */
    __global__ void prime(const uint lbl, const uint w, const uint h, const uint prev_w, const float *prev_msg, float *out) {
        const uint x = blockIdx.x * blockDim.x + threadIdx.x;
        const uint y = blockIdx.y * blockDim.y + threadIdx.y;

        // boundary check
        if (x >= w || y >= h) {
            return;
        }

        // initaise to the last layer's (x/2, y/2)
        float *target = ndx(lbl, w, out, x, y);
        const float *source = cndx(lbl, prev_w, prev_msg, x / 2, y / 2);

        for (uint i = 0; i < lbl; ++i) {
            target[i] = source[i];
        }
    }

    __global__ void get_results(const uint lbl, const uint w, const uint h, const float *u, const float *d, const float *l, const float *r, const float *pot, uchar *out) {
        const uint x = blockIdx.x * blockDim.x + threadIdx.x;
        const uint y = blockIdx.y * blockDim.y + threadIdx.y;

        // boundary check
        if (x < 1 || y < 1 || x >= w - 1|| y >= h - 1) {
            return;
        }

        uint min_label = 0;
        float min_value = HIP_MAX_NORMAL_F;

        const float *ut = cndx(lbl, w, u, x, y+1);
        const float *dt = cndx(lbl, w, d, x, y-1);
        const float *lt = cndx(lbl, w, l, x+1, y);
        const float *rt = cndx(lbl, w, r, x-1, y);
        const float *pott = cndx(lbl, w, pot, x, y);

        for (uint i = 0; i < lbl; ++i) {
            const float val = ut[i] + dt[i] + lt[i] + rt[i] + pott[i];
            if (val < min_value) {
                min_label = i;
                min_value = val;
            }
        }

        out[x + y * w] = min_label;
    }
}

std::vector<uchar> decode_hbp(const uchar labels, const uint layers, const uint max_iter, const uint width, const uint height, const std::vector<float> &pot, const float disc_trunc) {
    const uint tile_size = 16;
    dim3 block(tile_size, tile_size);

    // move the potentials to device memory
    float *dev_pot_initial;
    hipMalloc(&dev_pot_initial, pot.size() * sizeof(float));
    hipMemcpy(dev_pot_initial, &pot[0], pot.size() * sizeof(float), hipMemcpyHostToDevice);

    // pointers for the layers
    std::vector<float2> layer_sizes;
    std::vector<float *> dev_pot;

    // messages on the current layer and on the one below
    float *dev_u, *dev_d, *dev_l, *dev_r;
    float *dev_pu, *dev_pd, *dev_pl, *dev_pr;

    // inital set up
    dev_pot.push_back(dev_pot_initial);
    layer_sizes.push_back(make_float2(width, height));

    // create potentials for all layers
    for (uint i = 1; i < layers; ++i) {
        // calculate size of this layer
        const uint layer_width = (layer_sizes[i-1].x + 1) / 2, layer_height = (layer_sizes[i-1].y + 1) / 2;
        layer_sizes.push_back(make_float2(layer_width, layer_height));

        // memory for the potentials
        float *dev_layer_pot;
        hipMalloc(&dev_layer_pot, labels * layer_width * layer_height * sizeof(float));
        dev_pot.push_back(dev_layer_pot);

        // call the kernel to create the potential
        dim3 grid((layer_width + tile_size - 1) / tile_size, (layer_height + tile_size - 1) / tile_size);
        fill_next_layer_pot<<<grid, block>>>(labels, layer_width, layer_height, layer_sizes[i-1].x, layer_sizes[i-1].y, dev_pot[i-1], dev_layer_pot);
    }

    { // initalise memory for the messages
        const uint top_size = labels * width * height * sizeof(float);
        hipMalloc(&dev_u, top_size);
        hipMalloc(&dev_d, top_size);
        hipMalloc(&dev_l, top_size);
        hipMalloc(&dev_r, top_size);

        hipMalloc(&dev_pu, top_size);
        hipMalloc(&dev_pd, top_size);
        hipMalloc(&dev_pl, top_size);
        hipMalloc(&dev_pr, top_size);

        const uint elems = labels * layer_sizes.back().x * layer_sizes.back().y;
        const uint size = elems * sizeof(float);

        hipMemset(dev_u, 0, size);
        hipMemset(dev_d, 0, size);
        hipMemset(dev_l, 0, size);
        hipMemset(dev_r, 0, size);
    }

    { // run an initial round of BP on the bottom layer
        dim3 grid((layer_sizes.back().x + tile_size - 1) / tile_size, (layer_sizes.back().y + tile_size - 1) / tile_size);

        for (uint i = 0; i < max_iter; ++i) {
            bp<<<grid, block>>>(labels, layer_sizes.back().x, layer_sizes.back().y, disc_trunc, i, dev_pot.back(), dev_u, dev_d, dev_l, dev_r);
        }

        std::swap(dev_u, dev_pu);
        std::swap(dev_d, dev_pd);
        std::swap(dev_l, dev_pl);
        std::swap(dev_r, dev_pr);
    }

    // create messages using the messages on the layer below
    for (int i = layers - 2; i >= 0; --i) {
        dim3 grid((layer_sizes[i].x + tile_size - 1) / tile_size, (layer_sizes[i].y + tile_size - 1) / tile_size);
        prime<<<grid, block>>>(labels, layer_sizes[i].x, layer_sizes[i].y, layer_sizes[i+1].x, dev_pu, dev_u);
        prime<<<grid, block>>>(labels, layer_sizes[i].x, layer_sizes[i].y, layer_sizes[i+1].x, dev_pd, dev_d);
        prime<<<grid, block>>>(labels, layer_sizes[i].x, layer_sizes[i].y, layer_sizes[i+1].x, dev_pl, dev_l);
        prime<<<grid, block>>>(labels, layer_sizes[i].x, layer_sizes[i].y, layer_sizes[i+1].x, dev_pr, dev_r);

        // run the bp for this layer
        for (uint j = 0; j < max_iter; ++j) {
            bp<<<grid, block>>>(labels, layer_sizes[i].x, layer_sizes[i].y, disc_trunc, j, dev_pot[i], dev_u, dev_d, dev_l, dev_r);
        }

        std::swap(dev_u, dev_pu);
        std::swap(dev_d, dev_pd);
        std::swap(dev_l, dev_pl);
        std::swap(dev_r, dev_pr);
    }

    std::vector<uchar> results(width * height);
    { // collect results
        uchar *dev_out;
        hipMalloc(&dev_out, width * height * sizeof(uchar));

        dim3 grid((width + tile_size - 1) / tile_size, (height + tile_size - 1) / tile_size);
        get_results<<<grid, block>>>(labels, width, height, dev_pu, dev_pd, dev_pl, dev_pr, dev_pot_initial, dev_out);
        hipMemcpy(&results[0], dev_out, width * height * sizeof(uchar), hipMemcpyDeviceToHost);
        hipFree(dev_out);
    }

    { // clean up
        for (uint i = 0; i < dev_pot.size(); ++i) {
            hipFree(dev_pot[i]);
        }

        hipFree(dev_u);
        hipFree(dev_d);
        hipFree(dev_l);
        hipFree(dev_r);

        hipFree(dev_pu);
        hipFree(dev_pd);
        hipFree(dev_pl);
        hipFree(dev_pr);
    }

    return results;
}
